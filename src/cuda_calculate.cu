#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <iostream>
#include <thrust/complex.h>
#include "cuda_calculate.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include "thrust/system/cuda/detail/reduce.h"
#include "reduce.h"

#define GPUDEBUG

struct sub_f
{
    const double max;
    const double min;
    const double mean;
    const double width;


    sub_f(double _max, double _min, double _mean, double _width) : max(_max), min(_min), mean(_mean), width(_width)
    {}

    __device__
    double operator()(const double &x) const
    {
        double _x = x;
        if (x > max) {
            //    _x = 2 * max - x;
        }
        if (x < min) {
            //    _x = 2 * min - x;
        }
        double w = (width > 0) ? width : -width;
        return 1. / ((_x - mean) * (_x - mean) + 0.25 * w * w);
    }
};

struct sub_sigma
{
    const double sigma;

    sub_sigma(double _sigma) : sigma(_sigma)
    {}

    __device__
    double operator()(const double &x) const
    {
        return sqrt(15.)*3./sqrt(abs(x));
    }
};

struct sub_gauss
{
    const double t;

    sub_gauss(double _t) : t(_t)
    {}

    __device__
    double operator()(const double &x, const double &sigma) const
    {
//        thrust::complex<double> v(-0.5 / (sigma * sigma) * (x - t) * (x - t), 0);
//        return thrust::exp(v).real() / sigma;
        return exp(-0.5 / (sigma * sigma) * (x - t) * (x - t)) / sigma;
    }
};


thrust::device_vector<double> *d_t = nullptr;
thrust::device_vector<double> *d_sigma = nullptr;
thrust::device_vector<double> *d_x = nullptr;
thrust::device_vector<double> *d_w = nullptr;
//std::vector<double> *h_result = nullptr;

std::vector<double>
sub_cuda_normal_calculate(int bins, double min, double max, double x, double mean, double width, double f_min,
                          double f_max)
{
    std::vector<double> result;
    result.push_back(0);
#ifdef GPUDEBUG
    std::chrono::system_clock::time_point start, finish;
#endif

    if (d_t == nullptr) {
        d_t = new thrust::device_vector<double>(bins);
    }
    if (d_sigma == nullptr) {
        d_sigma = new thrust::device_vector<double>(bins);
    }
#ifdef GPUDEBUG
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::sequence((*d_t).begin(), (*d_t).end(), min + 0.5 * (max - min) / bins, (max - min) / bins);

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), sub_sigma(3));

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), (*d_sigma).begin(), sub_gauss(x));

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_t).begin(), sub_f(f_max, f_min, mean, width));

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), (*d_t).begin(), thrust::multiplies<double>());

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif

    result[0] = thrust::reduce((*d_t).begin(), (*d_t).end(), double(0), thrust::plus<double>());

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif

    return result;
}

void sub_cuda_gaus_prepare(std::vector<double> &x, std::vector<double> &w, int bins)
{
    if (d_x != nullptr) {
        delete d_x;
    }
    if (d_w != nullptr) {
        delete d_w;
    }
    d_x = new thrust::device_vector<double>(bins);
    d_w = new thrust::device_vector<double>(bins);
    thrust::copy(x.begin(), x.end(), (*d_x).begin());
    thrust::copy(w.begin(), w.end(), (*d_w).begin());
}

std::vector<double>
sub_cuda_gaus_calculate(int bins, double min, double max, double x, double mean, double width, double f_min,
                        double f_max)
{
    std::vector<double> result;
    result.push_back(0);
#ifdef GPUDEBUG
    std::chrono::system_clock::time_point start, finish;
#endif

    if (d_t == nullptr) {
        d_t = new thrust::device_vector<double>(bins);
    }
    if (d_sigma == nullptr) {
        d_sigma = new thrust::device_vector<double>(bins);
    }
#ifdef GPUDEBUG
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::copy((*d_x).begin(), (*d_x).end(), (*d_t).begin());

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), sub_sigma(3));

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), (*d_sigma).begin(), sub_gauss(x));

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_t).begin(), sub_f(f_max, f_min, mean, width));

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), (*d_t).begin(), thrust::multiplies<double>());
    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_w).begin(), (*d_t).begin(), thrust::multiplies<double>());

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif

    result[0] = thrust::reduce((*d_t).begin(), (*d_t).end(), double(0), thrust::plus<double>());

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif

    return result;
}

double cuda_normal_calculate(int bins, double min, double max, double x, double mean, double width, double f_min,
                             double f_max)
{
    //Malloc memory in GPU.
    thrust::device_vector<double> *d_t = new thrust::device_vector<double>(bins);
    thrust::device_vector<double> *d_sigma = new thrust::device_vector<double>(bins);

    //Prepare variable t from x_min to x_max.
    thrust::sequence((*d_t).begin(), (*d_t).end(), min + 0.5 * (max - min) / bins, (max - min) / bins);

    //Calculate sigma values from sigma kernel function sigma(t).
    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), sub_sigma(3));

    //Calculate gauss values form gauss kernel function gauss(x,t,sigma(t)).
    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), (*d_sigma).begin(), sub_gauss(x));

    //Calculate Breit-Wigner function f(t,mean,width) .
    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_t).begin(), sub_f(f_max, f_min, mean, width));

    //F(t) = f(t,mean,width)*gauss(x,t,sigma(t))
    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), (*d_t).begin(), thrust::multiplies<double>());

    //Sum(F(t))
    return thrust::reduce((*d_t).begin(), (*d_t).end(), double(0), thrust::plus<double>());
}

thrust::detail::temporary_array<double,
        thrust::iterator_system<thrust::detail::vector_base<double,
                thrust::device_malloc_allocator<double>>::iterator>::type> *tmp = nullptr;

#define device_vector_iterator thrust::detail::vector_base<double, thrust::device_malloc_allocator<double>>::iterator

template<typename BinaryFunction>
double myreduce(device_vector_iterator first,
                device_vector_iterator last,
                double init,
                BinaryFunction binary_op)
{
    using thrust::system::detail::generic::select_system;

    typedef typename thrust::iterator_system<device_vector_iterator>::type System;

    System system;
    auto input_system = thrust::detail::derived_cast(thrust::detail::strip_const(select_system(system)));
    if (tmp == nullptr) {
        tmp = new thrust::detail::temporary_array<double, System>(input_system, 100000);
    }

    //using thrust::system::detail::generic::reduce;
    using thrust::system::cuda::detail::reduce_detail::my_tuned_reduce;
    auto result = my_tuned_reduce(input_system, first, last, init,
                           binary_op, 100000, *tmp);
    return result;
}

std::vector<double>
sub_cuda_normal_calculate_tuned(int bins, double min, double max, double x, double mean, double width, double f_min,
                                double f_max)
{
    std::vector<double> result;
    result.push_back(0);
#ifdef GPUDEBUG
    std::chrono::system_clock::time_point start, finish;
#endif

    if (d_t == nullptr) {
        d_t = new thrust::device_vector<double>(bins);
    }
    if (d_sigma == nullptr) {
        d_sigma = new thrust::device_vector<double>(bins);
    }
#ifdef GPUDEBUG
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::sequence((*d_t).begin(), (*d_t).end(), min + 0.5 * (max - min) / bins, (max - min) / bins);

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), sub_sigma(3));

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), (*d_sigma).begin(), sub_gauss(x));

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_t).begin(), sub_f(f_max, f_min, mean, width));

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), (*d_t).begin(), thrust::multiplies<double>());

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif

    result[0] = myreduce((*d_t).begin(), (*d_t).end(), double(0), thrust::plus<double>());

#ifdef GPUDEBUG
    finish = std::chrono::high_resolution_clock::now();
    result.push_back(std::chrono::duration_cast<std::chrono::nanoseconds>(finish - start).count());
    start = std::chrono::high_resolution_clock::now();
#endif
    return result;
}

