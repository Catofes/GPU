#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <iostream>
#include <thrust/complex.h>
#include "cuda_calculate.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include "thrust/system/cuda/detail/reduce.h"

struct sub_f
{
    const double mean_1;
    const double mean_2;
    const double width_1;
    const double width_2;
    const double a;

    //Construct function
    sub_f(double _mean_1, double _mean_2, double _width_1, double _width_2, double _a) :
            mean_1(_mean_1), mean_2(_mean_2), width_1(_width_1), width_2(_width_2), a(_a)
    {}

    //Calculate function
    __device__
    double operator()(const double &x) const
    {
        double w1 = (width_1 > 0) ? width_1 : -width_1;
        double w2 = (width_2 > 0) ? width_2 : -width_2;
        double arg1 = x - mean_1;
        double arg2 = x - mean_2;
        return (1. / (arg1 * arg1 + 0.25 * w1 * w1) + a / (arg2 * arg2 + 0.25 * w2 * w2));
    }
};

struct sub_sigma
{
    sub_sigma()
    {}

    __device__
    double operator()(const double &x) const
    {
        return 10 / sqrt(x);
    }
};

struct sub_gauss
{
    const double t;

    sub_gauss(double _t) : t(_t)
    {}

    __device__
    double operator()(const double &x, const double &sigma) const
    {
//        thrust::complex<double> v(-0.5 / (sigma * sigma) * (x - t) * (x - t), 0);
//        return thrust::exp(v).real() / sigma;
        return exp(-0.5 / (sigma * sigma) * (x - t) * (x - t)) / sigma;
    }
};


thrust::device_vector<double> *d_t = nullptr;
thrust::device_vector<double> *d_sigma = nullptr;

double sub_cuda_normal_calculate(int bins,
                                 double x_min,
                                 double x_max,
                                 double x,
                                 double mean_1,
                                 double mean_2,
                                 double width_1,
                                 double width_2,
                                 double a)
{

    if (d_t == nullptr) {
        d_t = new thrust::device_vector<double>(bins);
    }
    if (d_sigma == nullptr) {
        d_sigma = new thrust::device_vector<double>(bins);
    }
    thrust::sequence((*d_t).begin(), (*d_t).end(), x_min + 0.5 * (x_max - x_min) / bins, (x_max - x_min) / bins);

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), sub_sigma());

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), (*d_sigma).begin(), sub_gauss(x));

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_t).begin(), sub_f(mean_1, mean_2, width_1, width_2, a));

    thrust::transform((*d_t).begin(), (*d_t).end(), (*d_sigma).begin(), (*d_t).begin(), thrust::multiplies<double>());

    return thrust::reduce((*d_t).begin(), (*d_t).end(), double(0), thrust::plus<double>());
}
